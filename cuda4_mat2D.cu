
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;
#define n 800

__global__ void matAdd(float (*)[n], float (*)[n],
					   float (*)[n]);

int main()
{
	const int memSize = sizeof(float) * n * n;
	// ��������� ������ ��� ���� ������
	float (*a)[n] = (float(*)[n])malloc(memSize);
	float (*b)[n] = (float(*)[n])malloc(memSize);
	float (*c)[n] = (float(*)[n])malloc(memSize);

		// ������������� ���� ���� ������
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a[i][j] = b[i][j] = 0.5;
			c[i][j] = 0;
		}
	}

	// �������� ������ ������
	float (*devA)[n];
	float (*devB)[n];
	float (*devC)[n];
	size_t pitch;

	// ��������� ������ �� ����������
	hipMallocPitch(&devA, &pitch, n * sizeof(float), n);
	hipMallocPitch(&devB, &pitch, n * sizeof(float), n);
	hipMallocPitch(&devC, &pitch, n * sizeof(float), n);

	hipMemcpy2D(devA, pitch, a, n * sizeof(float), n * sizeof(float), n, hipMemcpyHostToDevice);
	hipMemcpy2D(devB, pitch, b, n * sizeof(float), n * sizeof(float), n, hipMemcpyHostToDevice);

	dim3 numThreadsPerBlock(10, 10);
	dim3 numBlocks((n + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x,
					(n + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);
	matAdd<<<numBlocks, numThreadsPerBlock>>>(devA, devB, devC);

	hipMemcpy2D(c, n * sizeof(float), devC, pitch, n * sizeof(float), n, hipMemcpyDeviceToHost);

	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	for (int i = 0; i < 5; i++) {
		for (int j = 0; j < 5; j++) {
			cout << c[i][j] << ' ';
		}
		cout << '\n';
	}

	free(a);
	free(b);
	free(c);

	return 0;	
}

__global__ void matAdd(float (*A)[n], float (*B)[n],
					   float (*C)[n])
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	
	if (i < n && j < n) 
		C[i][j] = A[i][j] + B[i][j];
}






